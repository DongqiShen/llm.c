
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <float.h>


// cpu code reference

void attention_forward_cpu(float* out, float *preatt, float* att,
                        const float* inp,
                        int B, int T, int C, int NH) {
    // input is (B, T, 3C) Q, K, V
    // preattn, att are (B, NH, T, T)
    // output is (B, T, C)
    int C3 = 3 * C;
    int hs = C / NH;
    float scale = 1.0 / sqrt(hs);

    for (int b = 0; b < B; ++b) {
        for (int t = 0; t < T; ++t) { // 表示用于当作query的token
            for (int h = 0; h < NH; h++) {
                const float* query_t = inp + b * T * C3 + t * C3 + h * hs;
                float* preatt_bth = preatt + b * NH * T * T + h * T * T + t * T;
                float* att_bth = att + b * NH * T * T + h * T * T + t * T;

                // pass 1: calculate query dot key and maxval
                float maxval = -FLT_MAX;
                for (int t2 = 0; t2 <= t; ++t2) {
                    const float* key_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C;
                    // (query_t) dot (key_t2) 
                    float val = 0.0f;
                    for (int i = 0; i < hs; ++i) {
                        val += query_t[i] * key_t2[i];
                    }
                    val *= scale;
                    if (val > maxval) {
                        maxval = val;
                    }

                    preatt_bth[t2] = val;
                }
                // pad with -INFINITY outside of autoregressive region for debugging comparisons
                for (int t2 = t + 1; t2 < T; ++t2) {
                    preatt_bth[t2] = -INFINITY;
                }
                // pass 2: calculate the exp and keep track of sum
                float expsum = 0.0f;
                for (int t2 = 0; t2 <= t; ++t2) {
                    float expv = expf(preatt_bth[t2] - maxval);
                    expsum += expv;
                    att_bth[t2] = expv;
                }
                float expsum_inv = expsum == 0.0f ? 0.0f : 1.0f / expsum;

                // pass 3: normalize to get the softmax
                for (int t2 = 0; t2 <= T; ++t2) {
                    if (t2 <= t) {
                        att_bth[t2] *= expsum_inv;
                    } else {
                        // causal attention mask. not strictly necessary to set to zero here
                        // only doing this explicitly for debugging and checking to PyTorch
                        att_bth[t2] = 0;
                    }
                }

                // pass 4: accumulate weighted value into the output of attention
                float* out_bth = out + b * T * C + t * C + h * hs;
                for (int i = 0; i < hs; ++i) {
                    out_bth[i] = 0.0f;
                }
                for (int t2 = 0; t2 <= t; t2++) {
                    const float* value_t2 = inp + b * T * C3 + t2 * C3 + h * hs + 2 * C;
                    float att_btht2 = att_bth[t2];
                    for (int i = 0; i < hs; ++i) {
                        out_bth[i] += att_btht2 * value_t2[i];
                    }
                }
            }
        }
    }
}